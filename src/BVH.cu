#include "hip/hip_runtime.h"
#include "BVH.hpp"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <string.h>
#include <cstdlib>
#include <math.h>

void vectorDot(float& r, const Vector& a, const Vector& b){
  r = a.x * b.x + a.y * b.y + a.z * b.z;
}

void vectorCross(Vector& r, const Vector& a, const Vector& b){
  r.x = a.y * b.z - a.z * b.y;
  r.y = a.z * b.x - a.x * b.z;
  r.z = a.x * b.y - a.y * b.x;
}

void vectorAdd(Vector& r, const Vector& a, const Vector& b){
  r.x = a.x + b.x;
  r.y = a.y + b.y;
  r.z = a.z + b.z;
}

void vectorSub(Vector& r, const Vector& a, const Vector& b){
  r.x = a.x - b.x;
  r.y = a.y - b.y;
  r.z = a.z - b.z;
}

void vectorMul(Vector& r, const Vector&a, float s){
  r.x = a.x * s;
  r.y = a.y * s;
  r.z = a.z * s;
}

void vectorDiv(Vector& r, const Vector&a, float s){
  r.x = a.x / s;
  r.y = a.y / s;
  r.z = a.z / s;
}

void vectorNormalize(Vector& r, const Vector& a){
  float length;
  vectorDot(length, a, a);
  vectorDiv(r, a, sqrt(length));
}

float maxf(float a, float b){
  if(a>b){
	return a;
  }
  return b;
}


float minf(float a, float b){
  if(a<b){
	return a;
  }
  return b;
}

Mesh::Mesh(int nv, int ne, int nf){
  nVertices = nv;
  nEdges = ne;
  nFaces = nf;  

  vertices = new Vertex[nVertices];
  edges    = new Edge[nEdges];
  faces    = new Face[nFaces];
}

Mesh::~Mesh(){
  delete[] vertices;
  delete[] edges;
  delete[] faces;
}

void Mesh::perturbVertices(float magnitude, Vector& center){
  Vertex* displacement = new Vertex[nVertices];
  int*    count        = new int[nVertices];
  /*Reset displacement*/
  for(int i=0;i<nVertices;i++){
	displacement[i].x = 0;
	displacement[i].y = 0;
	displacement[i].z = 0;
	count[i]          = 0;
  }
  
  /*Displace faces along its distance vector from the center of the
	object*/
  for(int i=0;i<nFaces;i++){
	float perturbation;
	perturbation = magnitude*(rand()%RAND_MAX)/((float)RAND_MAX)-magnitude/2.25;

	Vector p1, p2, p3;
	Vector faceCenter;
	int* indices = faces[i].vertices;

	p1 = vertices[indices[0]];
	p2 = vertices[indices[1]];
	p3 = vertices[indices[2]];

	/*Compute center of face*/
	vectorAdd(faceCenter, p1, p2);
	vectorAdd(faceCenter, faceCenter, p3);
	vectorDiv(faceCenter, faceCenter, 3.0);

	vectorSub(faceCenter, faceCenter, center);

	vectorMul(faceCenter, faceCenter, perturbation);

	for(int j=0;j<3;j++){
	  vectorAdd(displacement[indices[j]], displacement[indices[j]], 
		faceCenter);
	  count[indices[j]]++;
	}
  }

  /*Displace vertices in model*/
  for(int i=0;i<nVertices;i++){
	vectorDiv(displacement[i], displacement[i], (float)count[i]);
	vectorAdd(vertices[i], displacement[i], vertices[i]);
  }

  delete[] displacement;
}

BVH::BVH(const char* fileName){
  mesh = 0;
  tree = 0;
  boxes = 0;
  adjacentFaces = 0;
  faceNodeMap = 0;

  std::ifstream in(fileName);
  
  if(in.is_open() ) {
	/*Read triangular mesh from file*/

	int n_vertices, n_edges, n_faces;
	
	in >> n_vertices >> n_edges >> n_faces;
	 
	mesh = new Mesh(n_vertices, n_edges, n_faces);
	
	for(int i=0;i<n_vertices;i++){
	  in >> mesh->vertices[i].x
	 >> mesh->vertices[i].y
	 >> mesh->vertices[i].z;
	}

	for(int i=0;i<n_edges;i++){
	  in >> mesh->edges[i].vertices[0] >> mesh->edges[i].vertices[1];
	}

	for(int i=0;i<n_faces;i++){
	  in >> mesh->faces[i].vertices[0]
	 >> mesh->faces[i].vertices[1]
	 >> mesh->faces[i].vertices[2]
	 >> mesh->faces[i].edges[0]
	 >> mesh->faces[i].edges[1]
	 >> mesh->faces[i].edges[2];
	}
	
	/*Read BVH*/
	in >> rootNode;
	in >> nNodes;
	in >> maxAdjacency;

	tree = new BHVNode[nNodes];
	boxes = new Box[nNodes];
	adjacentFaces = new int[nNodes * (maxAdjacency + 1)];

	faceNodeMap = new int[n_faces];

	/*Read nodes*/
	for(int i=0;i<nNodes;i++){
	  in >> tree[i].faceId 
	 >> tree[i].parent
	 >> tree[i].childs[0]
	 >> tree[i].childs[1]
	 >> tree[i].level;      
	  
	  if(tree[i].faceId != -1){
	faceNodeMap[tree[i].faceId] = i;
	  }
	}

	/*Read adjacency information*/
	int offset = maxAdjacency + 1;
	for(int i=0;i<nNodes;i++){
	  /*Read ammount of adjacent faces*/
	  in >> adjacentFaces[i*offset];

	  /*Read adjacent faces*/
	  for(int j=0;j<adjacentFaces[i*offset];j++){
	in >> adjacentFaces[i*offset+j+1];
	  }
	}

	/*Update BBox and visibility*/
	Vector displacement;
	displacement.x = displacement.y = displacement.z = 0;

	update(displacement);
  }else{
	std::cout << "Could not open file:: " << fileName << std::endl;
  }
}

BVH::~BVH(){
  if(mesh){
	delete mesh;
  }
  if(tree){
	delete[] tree;
  }
  if(adjacentFaces){
	delete[] adjacentFaces;
  }
  if(faceNodeMap){
	delete[] faceNodeMap;
  }
  if(boxes){
	delete[] boxes;
  }
}

Box::Box(){
  reset();
}

bool Box::intersects(const Box& b)const{
  if(!( (b.min.x <= max.x) && (b.max.x >= min.x) ))
	return false;
  
  if(!( (b.min.y <= max.y) && (b.max.y >= min.y) ))
	return false;
  
  if(!( (b.min.z <= max.z) && (b.max.z >= min.z) ))
	return false;
  
  return true;
}

void Box::reset(){
  min.x = min.y = min.z =  1E100;
  max.x = max.y = max.z = -1E100;
}

void Box::addPoint(Vector& p){
  if(min.x > p.x){
	min.x = p.x;
  }
  if(max.x < p.x){
	max.x = p.x;
  }

  if(min.y > p.y){
	min.y = p.y;
  }
  if(max.y < p.y){
	max.y = p.y;
  }

  if(min.z > p.z){
	min.z = p.z;
  }
  if(max.z < p.z){
	max.z = p.z;
  }
}

int binarySearch(int* A, int key, int imin, int imax){
  while(imax >= imin){
	int imid = imin + ((imax - imin) / 2);

	if(A[imid] < key){
	  imin = imid + 1;
	}else if(A[imid] > key){
	  imax = imid - 1;
	}else{
	  return imid;
	}
  }

  return -1;
}

void BVH::update(const Vector& displacement){
  /*Update all bounding boxes given a displacement*/
  updateNodeBBox(rootNode, displacement);

  /*Update visibility-bits*/
  updateNodeVisibility(rootNode);
}

/*Recursively update bounding-boxes*/
void BVH::updateNodeBBox(int curNode, const Vector& displacement){
  if(curNode == -1){
	return;
  }
  
  if(tree[curNode].faceId == -1){
	/*First update bounding box of children*/
	
	updateNodeBBox(tree[curNode].childs[0], displacement);
	updateNodeBBox(tree[curNode].childs[1], displacement);
	
	/*Add min and max points of child boxes to current box*/
	boxes[curNode].reset();
	int* childs = tree[curNode].childs; 
	if(childs[0] != -1){
	  boxes[curNode].addPoint(boxes[childs[0]].min);
	  boxes[curNode].addPoint(boxes[childs[0]].max);
	}
	
	if(childs[1] != -1){
	  boxes[curNode].addPoint(boxes[childs[1]].min);
	  boxes[curNode].addPoint(boxes[childs[1]].max);
	}
  }else{
	/*Update the bounding box according to the stored geometry*/
	boxes[curNode].reset();
	int* vertices = mesh->faces[tree[curNode].faceId].vertices;
	boxes[curNode].addPoint(mesh->vertices[vertices[0]]);
	boxes[curNode].addPoint(mesh->vertices[vertices[1]]);
	boxes[curNode].addPoint(mesh->vertices[vertices[2]]);

	/*Add displacement*/
	Vector p1, p2;
	vectorAdd(p1, boxes[curNode].min, displacement);
	vectorAdd(p2, boxes[curNode].max, displacement);

	boxes[curNode].addPoint(p1);
	boxes[curNode].addPoint(p2);
  }  
}

/*Recursively update visibility-bits*/
void BVH::updateNodeVisibility(int curNode){
  if(curNode == -1){
	return;
  }

  tree[curNode].visibility = 0;

  if(tree[curNode].faceId != -1){
	/*Determine visibility of a triangle*/

	int* vertices = mesh->faces[tree[curNode].faceId].vertices;

	Vector faceNormal;
	Vector ba;
	Vector ca;

	vectorSub(ba, mesh->vertices[vertices[1]], mesh->vertices[vertices[0]]);
	vectorSub(ca, mesh->vertices[vertices[2]], mesh->vertices[vertices[0]]);

	vectorCross(faceNormal, ba, ca);

	for(int i=0;i<3;i++){
	  for(int j=0;j<3;j++){
	for(int k=0;k<3;k++){
	  if((i == 1) && (j == 1) && (k == 1)){
		/*Skip, center*/
	  }else{
		Vector testVector;
		testVector.x = 1-i;
		testVector.y = 1-j;
		testVector.z = 1-k;

		float dot;
		vectorDot(dot, testVector, faceNormal);

		if(dot >= 0){
		  tree[curNode].visibility |= 1 << (i*9+j*3+k);
		}
	  }
	}
	  }
	}
  }else{
	/*Determine visibility by AND-ing visibility-bits of childs*/

	int* childs = tree[curNode].childs;
	updateNodeVisibility(childs[0]);
	updateNodeVisibility(childs[1]);

	if(childs[0] != -1 && childs[1] != -1){
	  tree[curNode].visibility = (tree[childs[0]].visibility &
				  tree[childs[1]].visibility);
	}else{
	  if(childs[0] != -1){
	tree[curNode].visibility = tree[childs[0]].visibility;
	  }
	  if(childs[1] != -1){
	tree[curNode].visibility = tree[childs[1]].visibility;
	  }
	}
  }
}

bool BVH::checkAdjacency(int node, int query){
  int index = binarySearch(adjacentFaces + node*(maxAdjacency+1), query, 
			   1, adjacentFaces[node*(maxAdjacency+1)] + 1);

  if(index == -1){
	return false;
  }
  /*Query-node is adjacent to node*/
  return true;
}

/*Finds all nodes in the tree that may intersect with the given face*/
bool BVH::checkCollisionsFace(const int curNode, const int queryNode, 
				  Collisions* candidates){

  /*A face can't collide with itself*/
  if(curNode == queryNode){
	return false;
  }

  /*Check if the query is valid*/
  if(tree[queryNode].faceId == -1){
	return false;
  }
  
  if(boxes[curNode].intersects(boxes[queryNode])){
	/*Possible collision, since bounding boxes intersect*/

	int child = queryNode;

	/*Traverse the tree to the root until the current node has no
	  parent*/
	while(true){
	  if(child == curNode){
	if(tree[curNode].visibility){
	  return false;
	}
	break;
	  }
	  if(child == -1){
	/*child is the root of an object (in case of multiple objects,
	  this is not the root of the tree)*/
	break;
	  }
	  if(tree[curNode].level == tree[child].level){
	if(checkAdjacency(curNode, child)){
	  if(tree[curNode].visibility & tree[child].visibility){
		return false;
	  }
	}else{
	  /*Nodes are not adjacent*/
	}
	break;
	  }

	  child = tree[child].parent;
	}
	
	if(tree[curNode].faceId != -1){
	  /*Store result in candidates*/
	  candidates->storeFaceFaceResult(tree[queryNode].faceId, 
					  tree[curNode].faceId); 
	}else{
	  bool c1 = false;
	  bool c2 = false;
	  if(tree[curNode].childs[0] != -1){
	c1 = checkCollisionsFace(tree[curNode].childs[0], queryNode, 
				 candidates);
	  }
	  if(tree[curNode].childs[1] != -1){
	c2 = checkCollisionsFace(tree[curNode].childs[1], queryNode, 
				 candidates);
	  }
	  return c1 || c2;
	}
  }
  return false;
}

void BVH::checkFaceFaceCollisions(Collisions* c){
  for(int i=0;i<mesh->nFaces;i++){ 
	int queryNode = faceNodeMap[i];
	checkCollisionsFace(rootNode, queryNode, c); 
  }
}

void BVH::checkCollisions(Collisions* c, const Vector& displacement){
  /*Initialize / clear the stored collisions*/
  c->initialize();
  
  checkFaceFaceCollisions(c);

  c->breakDown(this, displacement);
}

Collisions::Collisions(const BVH* b, int max){
  maxSize = max;

  nFaces    = b->mesh->nFaces;
  nVertices = b->mesh->nVertices;
  nEdges    = b->mesh->nEdges;

  if(b->mesh){  
	potentialFaceFace     = new int[nFaces * maxSize];
	nPotentialFaces       = new int[nFaces];

	potentialVertexFace   = new int[nVertices * maxSize];
	nPotentialVertexFaces = new int[nVertices];
	
	potentialEdgeEdge     = new int[nEdges * maxSize];
	nPotentialEdgeEdges   = new int[nEdges];
  }else{
	potentialFaceFace     = 0;
	nPotentialFaces       = 0;

	potentialVertexFace   = 0;
	nPotentialVertexFaces = 0;

	potentialEdgeEdge     = 0;
	nPotentialEdgeEdges   = 0;
  }
}

Collisions::~Collisions(){
  if(potentialFaceFace){
	delete[] potentialFaceFace;
	delete[] nPotentialFaces;
	delete[] potentialVertexFace;
	delete[] nPotentialVertexFaces;
	delete[] potentialEdgeEdge;
	delete[] nPotentialEdgeEdges;
  }
}

/*Resets all collisions*/
void Collisions::initialize(){  
  for(int i=0;i<nFaces;i++){
	nPotentialFaces[i] = 0;
  }
  
  for(int i=0;i<nVertices;i++){
	nPotentialVertexFaces[i] = 0;
  }

  for(int i=0;i<nEdges;i++){
	nPotentialEdgeEdges[i] = 0;
  }
}

void Collisions::storeFaceFaceResult(int f1, int f2){
  int index = nPotentialFaces[f1];

  if(index >= maxSize){
	/*Array is to small to store the results*/
	//extendStorage();
	std::cout << "maxSize is too small for storing all results" << std::endl;
  }else{
	potentialFaceFace[f1*maxSize + index] = f2;
	nPotentialFaces[f1]++;
  }
}

void Collisions::storeVertexFaceResult(int v, int f){
  int index = nPotentialVertexFaces[v];

  if(index >= maxSize){
	/*Array is to small to store the results*/
	//extendStorage();
	std::cout << "maxSize is too small for storing all results" << std::endl;
  }else{  
	potentialVertexFace[v*maxSize + index] = f;
	nPotentialVertexFaces[v]++;
  }
}

void Collisions::storeEdgeEdgeResult(int e1, int e2){
  int index = nPotentialEdgeEdges[e1];

  if(index >= maxSize){
	/*Array is to small to store the results*/
	//extendStorage();
	std::cout << "maxSize is too small for storing all results" << std::endl;
  }else{ 
	potentialEdgeEdge[e1*maxSize + index] = e2;
	nPotentialEdgeEdges[e1]++;
  }
}


__global__ void breakDownDeel1(int nFaces, int maxSize, int* nPotFace, int* potFaceFace, int* VFOutput, int* EEOutput)
{
	int faceA = (blockDim.x * blockIdx.x + threadIdx.x);
	int j = (blockDim.y * blockIdx.y + threadIdx.y);
    if (faceA < nFaces) {
		int nPairs = nPotFace[faceA];
		if(j < nPairs) {
			faceB = potFaceFace[faceA * maxSize + j];
			
			// doe berekeningen hier
		}
	}   
}

void Collisions::breakDown(const BVH* bvh, const Vector& displacement){  
	
	int N = 3;
    size_t size = N * sizeof(int);

    // Allocate input vectors h_A and h_B in host memory
    int* h_A = (int*)malloc(size);
    int* h_B = (int*)malloc(size);
	int* h_C = (int*)malloc(size);

    // Initialize input vectors
    for(int i=0; i<3; i++) {
		h_A[i] = 2*i;
		h_B[i] = 3*(i+1);
	}

    // Allocate vectors in device memory
    int* d_A;
    hipMalloc(&d_A, size);
    int* d_B;
    hipMalloc(&d_B, size);
    int* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	cudaPrintfInit();
	
    // Invoke kernel
    int threadsPerBlock = 2;
    int blocksPerGrid = N;
    breakDownDeel1<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	int threadsPerBlock = 1;
	int blocksPerGrid = nFaces;
	breakDownDeel1<<<blocksPerGrid, threadsPerBlock>>>(this, bvh, displacement, nFaces);
}

/*Not used anymore*/
void Collisions::symmetrize(){
  /*In order to symmetrize, sort potentialFaceFace and make a copy of
	nPotentialFaceFaces.*/
  int* tmpNPotentialFaceFaces = new int[nFaces];

  memcpy(tmpNPotentialFaceFaces, nPotentialFaces, 
	 sizeof(int) * nFaces);

  /*Sort each array*/
  for(int i=0;i<nFaces;i++){
	std::sort(potentialFaceFace + i*maxSize, 
		  potentialFaceFace + i*maxSize + tmpNPotentialFaceFaces[i]); 
  }

  for(int faceA=0;faceA<nFaces;faceA++){
	for(int indexB=0;indexB<nPotentialFaces[faceA];indexB++){
	  int faceB = potentialFaceFace[faceA * maxSize + indexB];
	  
	  /*Check if faceA exists in array of faceB*/
	  /*We use tmpNPotentialFaceFaces here, since array
	potentialFaceFace will be updated after the part which is
	sorted. The copy just contains the number of elements in the
	old situation, which are sorted.*/
	  int indexA = binarySearch(potentialFaceFace + faceB*maxSize,
				faceA, 0, tmpNPotentialFaceFaces[faceB]);

	  if(indexA == -1){
	/*Face A is not found in the array of faceB*/
	storeFaceFaceResult(faceB, faceA); //Also updates nPotentialFaces
	  }
	}
  }

  delete[] tmpNPotentialFaceFaces;
}

void Collisions::extendStorage(){
  int newMaxSize = maxSize*2;

  int* tmppotentialFaceFace     = new int[nFaces * newMaxSize];
  int* tmppotentialVertexFace   = new int[nVertices * newMaxSize];    
  int* tmppotentialEdgeEdge     = new int[nEdges * newMaxSize];

  /*Copy data*/
  for(int i=0;i<nFaces;i++){
	memcpy(tmppotentialFaceFace + i * newMaxSize, 
	   potentialFaceFace + i * maxSize, sizeof(int)*maxSize);
  }

  for(int i=0;i<nVertices;i++){
	memcpy(tmppotentialVertexFace + i * newMaxSize, 
	   potentialVertexFace + i * maxSize, sizeof(int)*maxSize);
  }

  for(int i=0;i<nEdges;i++){
	memcpy(tmppotentialEdgeEdge + i * newMaxSize, 
	   potentialEdgeEdge + i * maxSize, sizeof(int)*maxSize);
  }

  maxSize = newMaxSize;
  
  delete[] potentialFaceFace;
  delete[] potentialVertexFace;
  delete[] potentialEdgeEdge;

  potentialFaceFace = tmppotentialFaceFace;
  potentialVertexFace = tmppotentialVertexFace;
  potentialEdgeEdge = tmppotentialEdgeEdge;
}
